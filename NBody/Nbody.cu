#include <iostream>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <ctime>
#include <cmath>

#define TIMESTEPS 100
#define PARTICLE_COUNT 10
#define DIM 2
#define G 6.673 * powf(10, -11)

using namespace std;


__device__ float distance(float x1, float y1, float x2, float y2)
{
	return sqrtf(powf((x1 - x2), 2) + powf((y1 - y2), 2));
}

__device__ float cube(float num){
	return powf(num, 3);
}

__global__ void myprint(){
	return;
}

__global__ void initialize(float *pos_x, float* pos_y, float* masses, float* velocities_x, float* velocities_y){
	hiprandState_t state;
	hiprand_init(0, 0, 0, &state);
	pos_x[threadIdx.x] = float(hiprand(&state) % 101 + (-50));
	pos_y[threadIdx.x] = float(hiprand(&state) % 101 + (-50));
	velocities_x[threadIdx.x] = float(hiprand(&state) % 11 + (-5)) / 1000.0;
	velocities_y[threadIdx.x] = float(hiprand(&state) % 11 + (-5)) / 1000.0;
	masses[threadIdx.x] = float(hiprand(&state) % 10000);
}
__global__ void updateVelocities(float *masses, float* velocities_x, float* velocities_y, float* pos_x, float* pos_y)
{
	int q = threadIdx.x;
	int k = threadIdx.y;
	if (q != k && q < PARTICLE_COUNT && k < PARTICLE_COUNT){
		velocities_x[q] -= (G * masses[q] * masses[k]) / cube(distance(pos_x[q], pos_y[q], pos_x[k], pos_y[k])) * (pos_x[q] - pos_x[k]);
		velocities_y[q] -= (G * masses[q] * masses[k]) / cube(distance(pos_x[q], pos_y[q], pos_x[k], pos_y[k])) * (pos_y[q] - pos_y[k]);
		return;
	}
}
__global__ void updatePositions(float *pos_x, float *pos_y, float* velocities_x, float* velocities_y){
	if (threadIdx.x < PARTICLE_COUNT){
		pos_x[threadIdx.x] += velocities_x[threadIdx.x];
		pos_y[threadIdx.x] += velocities_y[threadIdx.x];
	}
}

int main() {
	srand(1);
	dim3 threadsperblock(PARTICLE_COUNT, PARTICLE_COUNT);
	// Initialize the host variables
	float ih_pos_x[PARTICLE_COUNT];
	float ih_pos_y[PARTICLE_COUNT];
	float fh_pos_x[PARTICLE_COUNT];
	float fh_pos_y[PARTICLE_COUNT];

	float h_masses[PARTICLE_COUNT];
	
	// Initialize the device variables
	float d_pos_x[PARTICLE_COUNT];
	float d_pos_y[PARTICLE_COUNT];
	float d_masses[PARTICLE_COUNT];
	float d_velocities_x[PARTICLE_COUNT];
	float d_velocities_y[PARTICLE_COUNT];
	float d_distances_x[PARTICLE_COUNT];
	float d_distances_y[PARTICLE_COUNT];

	if ( 
		(hipMalloc((void**)&d_pos_x, sizeof(float) * PARTICLE_COUNT) != hipSuccess) ||
		(hipMalloc((void**)&d_pos_y, sizeof(float) * PARTICLE_COUNT) != hipSuccess) ||
		(hipMalloc((void**)&d_masses, sizeof(float) * PARTICLE_COUNT) != hipSuccess) ||
		(hipMalloc((void**)&d_velocities_x, sizeof(float) * PARTICLE_COUNT) != hipSuccess) ||
		(hipMalloc((void**)&d_velocities_y, sizeof(float) * PARTICLE_COUNT) != hipSuccess) ||
		(hipMalloc((void**)&d_distances_x, sizeof(float) * PARTICLE_COUNT) != hipSuccess) ||
		(hipMalloc((void**)&d_distances_y, sizeof(float) * PARTICLE_COUNT) != hipSuccess) )

	{
		cout << "Error: Allocating memory" << endl;
		return 1;
	}
	initialize<<<PARTICLE_COUNT / 256 + 1, 256>>>(d_pos_x, d_pos_y, d_masses, d_velocities_x, d_velocities_y);
	hipMemcpy(ih_pos_x, d_pos_x, sizeof(float) * PARTICLE_COUNT, hipMemcpyDeviceToHost);
	hipMemcpy(ih_pos_y, d_pos_y, sizeof(float) * PARTICLE_COUNT, hipMemcpyDeviceToHost);
	hipMemcpy(d_masses, h_masses, sizeof(float) * PARTICLE_COUNT, hipMemcpyDeviceToHost);

	for (int i = 0; i < TIMESTEPS; ++i){
		updateVelocities<<<PARTICLE_COUNT / 256 + 1, threadsperblock>>>(d_pos_x, d_pos_y, d_masses, d_velocities_x, d_velocities_y);
		updatePositions<<<PARTICLE_COUNT / 256 + 1, 256 >>>(d_pos_x, d_pos_y, d_velocities_x, d_velocities_y);
	}
	hipMemcpy(fh_pos_x, d_pos_x, sizeof(float) * PARTICLE_COUNT, hipMemcpyDeviceToHost);
	hipMemcpy(fh_pos_y, d_pos_y, sizeof(float) * PARTICLE_COUNT, hipMemcpyDeviceToHost);
	
	cout << "INITIAL POSITIONS" << endl;
	
	for (int i = 0; i < PARTICLE_COUNT; ++i){
		cout << i << ":\t" << h_masses[i] << "\t" << ih_pos_x[i] << ", " << ih_pos_y[i] << endl;
	}
	cout << "FINAL POSITIONS" << endl;
	for (int j = 0; j < PARTICLE_COUNT; ++j){
		cout << j << ":\t" << h_masses[j] << "\t" << ih_pos_x[j] << ", " << ih_pos_y[j] << endl;
	}
	getchar();


}