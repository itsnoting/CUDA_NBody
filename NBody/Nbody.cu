#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <ctime>
#include <cmath>

#define TIMESTEPS 100
#define PARTICLE_COUNT 10
#define DIM 2
#define G 6.673 * pow(10, -11)

using namespace std;


__device__ float distance(float x1, float y1, float x2, float y2)
{
	return sqrt(pow((x1 - x2), 2) + pow((y1 - y2), 2));
}

__global__ void initialize(float *pos_x, float* pos_y, float* masses, float* velocities_x, float* velocities_y){
	hiprandState_t state;
	hiprand_init(0, 0, 0, &state);
	pos_x[threadIdx.x] = float(hiprand(&state) % 101 + (-50));
	pos_y[threadIdx.x] = float(hiprand(&state) % 101 + (-50));
	velocities_x[threadIdx.x] = float(hiprand(&state) % 11 + (-5)) / 1000.0;
	velocities_y[threadIdx.x] = float(hiprand(&state) % 11 + (-5)) / 1000.0;
	masses[threadIdx.x] = float(hiprand(&state) % 10000);
}
__global__ void updateVelocities(float *masses, float* velocities_x, float* velocities_y, float* pos_x, float* pos_y)
{
	velocities_x[threadIdx.x] -= (G * masses[threadIdx.x] * masses[threadIdx.y]) / 
		pow(distance(pos_x[threadIdx.x], pos_y[threadIdx.x], pos_x[threadIdx.y], pos_y[threadIdx.y]), 3) * 
		(pos_x[threadIdx.x] - pos_x[threadIdx.y]);
	velocities_y[threadIdx.x] -= (G * masses[threadIdx.x] * masses[threadIdx.y]) / 
		pow(distance(pos_x[threadIdx.x], pos_y[threadIdx.x], pos_x[threadIdx.y], pos_y[threadIdx.y]), 3) * 
		(pos_y[threadIdx.x] - pos_y[threadIdx.y]);

}
__global__ void updatePositions(float *pos_x, float *pos_y, float* masses, float* velocities_x, float* velocities_y){
	
}

int main() {
	srand(1);
	dim3 threadsperblock(PARTICLE_COUNT, DIM);
	// Initialize the host variables
	float h_pos_x[PARTICLE_COUNT]= {};
	float h_pos_y[PARTICLE_COUNT] = {};
	float h_masses[PARTICLE_COUNT] = {};
	float h_velocities_x[PARTICLE_COUNT] = {};
	float h_velocities_y[PARTICLE_COUNT] = {};
	float h_distances_x[PARTICLE_COUNT] = {};
	float h_distances_y[PARTICLE_COUNT] = {};

	// Initialize the device variables
	float d_pos_x[PARTICLE_COUNT] = {};
	float d_pos_y[PARTICLE_COUNT] = {};
	float d_masses[PARTICLE_COUNT] = {};
	float d_velocities_x[PARTICLE_COUNT] = {};
	float d_velocities_y[PARTICLE_COUNT] = {};
	float d_distances_x[PARTICLE_COUNT] = {};
	float d_distances_y[PARTICLE_COUNT] = {};

	if ( 
		(hipMalloc((void**)&d_pos_x, sizeof(float) * PARTICLE_COUNT) != hipSuccess) ||
		(hipMalloc((void**)&d_pos_y, sizeof(float) * PARTICLE_COUNT) != hipSuccess) ||
		(hipMalloc((void**)&d_masses, sizeof(float) * PARTICLE_COUNT) != hipSuccess) ||
		(hipMalloc((void**)&d_velocities_x, sizeof(float) * PARTICLE_COUNT * DIM) != hipSuccess) ||
		(hipMalloc((void**)&d_velocities_y, sizeof(float) * PARTICLE_COUNT * DIM) != hipSuccess) ||
		(hipMalloc((void**)&d_distances_x, sizeof(float) * PARTICLE_COUNT * DIM) != hipSuccess) ||
		(hipMalloc((void**)&d_distances_y, sizeof(float) * PARTICLE_COUNT * DIM) != hipSuccess) )

	{
		cout << "Error: Allocating memory" << endl;
		return 1;
	}
	initialize << < PARTICLE_COUNT / 256 + 1, 256 >> >(d_pos_x, d_pos_y, d_masses, d_velocities_x, d_velocities_y);
	for (int i = 0; i < TIMESTEPS; ++i){
		updatePositions << <PARTICLE_COUNT / 256 + 1, 256 >> >(d_pos_x, d_pos_y, d_masses, d_velocities_x, d_velocities_y);
		continue;
	}

}